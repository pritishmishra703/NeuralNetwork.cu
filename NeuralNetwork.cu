
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <algorithm>

#define BATCH_SIZE 4

std::vector<std::string> split(const std::string& str, char delimiter) {
    std::vector<std::string> tokens;
    std::string token;
    std::istringstream tokenStream(str);
    while (std::getline(tokenStream, token, delimiter)) {
        tokens.push_back(token);
    }
    return tokens;
}

void loadCSV(const std::string& filename, std::vector<std::vector<float>>& X, std::vector<float>& Y) {
    std::ifstream file(filename);
    std::string line;
    
    while (std::getline(file, line)) {
        std::vector<std::string> tokens = split(line, ',');
        std::vector<float> x_row;
        
        for (size_t i = 0; i < tokens.size() - 1; ++i) {
            x_row.push_back(std::stof(tokens[i]));
        }
        
        X.push_back(x_row);
        Y.push_back(std::stof(tokens.back()));
    }
}

void createBatches(const std::vector<std::vector<float>>& X, const std::vector<float>& Y, int batchSize) {
    size_t total_samples = X.size();
    
    for (size_t i = 0; i < total_samples; i += batchSize) {
        size_t batch_end = std::min(i + batchSize, total_samples);

        std::cout << "Batch from index " << i << " to " << batch_end - 1 << std::endl;
        std::cout << "X Batch:" << std::endl;
        for (size_t j = i; j < batch_end; ++j) {
            for (float val : X[j]) {
                std::cout << val << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "Y Batch:" << std::endl;
        for (size_t j = i; j < batch_end; ++j) {
            std::cout << Y[j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    std::vector<std::vector<float>> X;
    std::vector<float> Y;
    
    // Load CSV
    loadCSV("data/sample.csv", X, Y);

    // Print the data for debugging
    std::cout << "Loaded data successfully." << std::endl;
    std::cout << "Total samples: " << X.size() << std::endl;
    std::cout << "Creating batches of size: " << BATCH_SIZE << std::endl;

    // Create and print batches
    createBatches(X, Y, BATCH_SIZE);
    
    return 0;
}
