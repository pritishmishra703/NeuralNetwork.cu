
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <algorithm>

// Structure to hold our dataset
struct Dataset {
    std::vector<std::vector<float>> X;
    std::vector<float> y;
};

// Function to load CSV file
Dataset loadCSV(const std::string& filename) {
    Dataset data;
    std::ifstream file(filename);
    std::string line;

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string value;
        std::vector<float> row;

        while (std::getline(iss, value, ',')) {
            row.push_back(std::stof(value));
        }

        // Last column is y, rest are X
        data.y.push_back(row.back());
        row.pop_back();
        data.X.push_back(row);
    }

    return data;
}

// Function to create batches
std::vector<Dataset> createBatches(const Dataset& data, int batchSize) {
    std::vector<Dataset> batches;
    int numSamples = data.X.size();
    int numBatches = (numSamples + batchSize - 1) / batchSize;

    for (int i = 0; i < numBatches; ++i) {
        Dataset batch;
        int start = i * batchSize;
        int end = std::min(start + batchSize, numSamples);

        batch.X.assign(data.X.begin() + start, data.X.begin() + end);
        batch.y.assign(data.y.begin() + start, data.y.begin() + end);

        batches.push_back(batch);
    }

    return batches;
}

// Function to print batches (for debugging)
void printBatches(const std::vector<Dataset>& batches) {
    for (size_t i = 0; i < batches.size(); ++i) {
        std::cout << "Batch " << i + 1 << ":\n";
        for (size_t j = 0; j < batches[i].X.size(); ++j) {
            std::cout << "  Sample " << j + 1 << ": ";
            for (float val : batches[i].X[j]) {
                std::cout << val << " ";
            }
            std::cout << "| " << batches[i].y[j] << "\n";
        }
        std::cout << "\n";
    }
}

int main() {
    // Load the CSV file
    Dataset data = loadCSV("data/sample.csv");

    // Create batches
    std::vector<Dataset> batches = createBatches(data, 3); // Batch size of 32

    // Print batches for debugging
    printBatches(batches);

    return 0;
}
